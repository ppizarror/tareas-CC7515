﻿/** Instrucciones
 *
 * El juego comienza con una configuracion al azar entre celdas vivas y muertas.
 *
 * Para modificar los valores de la ejecucion simplemente hay que modificar los
 * valores de las constantes declaradas mas abajo.
 *
 * N: Numero de filas que tendra la matriz que almacene el estado del juego.
 * M: Numero de columnas que tendra la matriz que almacene el estado del juego.
 *
 * BLOCK_SIZE: cantidad de threads que tendra cada bloque.
 * SRAND_VALUE: semilla que se ocupara para generar los numeros al azar.
 * GOLIF: Indicador en caso de que se quiera verificar la cantidad de celdas
 *        vecinas vivas usando solo IF's.
 * IMPRIMIR: Indicador en caso de que se necesite imprimir las matrices (esto
 *           afecta considerablemente el rendimiento de la solucion)
 */

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <ctime>

 /* Declaración de constantes */
#define N 20 // Filas
#define M 20 // Columnas

#define BLOCK_SIZE 128   // Tamaño de bloque
#define SRAND_VALUE 1998 // Semilla para generar numeros random
#define GOLIF 0          // Ejecutar el juego preguntando con IF (0:Falso 1:Verdadero)
#define IMPRIMIR 0       // Imprimir o no las matrices de entrada y de salida

/* Declaración de funciones */
__global__ void GOL(int dimFilas, int dimColumnas, int *grid, int *newGrid);

__global__ void ghostRows(int dimFilas, int dimColumnas, int *grid);

__global__ void ghostCols(int dimFilas, int dimColumnas, int *grid);

__global__ void GOL_IF(int dimFilas, int dimColumnas, int *grid, int *newGrid);

void imprimir(int *matriz);

/* Método principal */
int main(int argc, char *argv[]) {

	int i, j;
	int *h_grid; // Matriz en CPU
	int *d_grid; // Matriz en GPU
	int *d_newGrid; // Matriz auxiliar usada solo en GPU
	int *d_tmpGrid; // Puntero auxiliar para cambiar las matrices

	signed t0, t1; // Variables para medir tiempo
	double time = 0; //variables para medir tiempo

	double Noperaciones = 0; // Variable para medir cantidad de operaciones ejecutadas

	int dimFilas = N; // Dimensiones del juego de la vida (Filas), sin contar las filas fantasmas
	int dimColumnas = M; // Dimensiones del juego de la vida (Columnas), sin contar las columnas fantasmas

	size_t bytes = sizeof(int) * (dimFilas + 2) *
		(dimColumnas + 2);// Se annade mas espacio para dejar filas y columnas fantasmas

// Solicitamos memoria para la matriz en la CPU
	h_grid = (int *)malloc(bytes);

	// Solicitamos memoria para las matrices en la GPU
	hipMalloc(&d_grid, bytes);
	hipMalloc(&d_newGrid, bytes);

	// Colocamos valores aleatorios en la matriz inicialmente
	srand(SRAND_VALUE);
	for (i = 1; i <= dimFilas; i++) {
		for (j = 1; j <= dimColumnas; j++) {
			h_grid[i * (dimColumnas + 2) + j] = rand() % 2;
		}
	}

	// Copiamos valores iniciales de la matriz a la GPU
	hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_newGrid, h_grid, bytes, hipMemcpyHostToDevice);

	// Establecemos los tamannos de los bloques y la cantidad de bloques a utilizar
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
	int linGrid = (int)ceil((dimFilas * dimColumnas) / (float)(BLOCK_SIZE * BLOCK_SIZE));
	dim3 gridSize(linGrid, linGrid, 1);

	dim3 cpyBlockSize(BLOCK_SIZE, 1, 1);
	dim3 cpyGridRowsGridSize((int)ceil(dimFilas / (float)cpyBlockSize.x), 1, 1);
	dim3 cpyGridColsGridSize((int)ceil((dimColumnas + 2) / (float)cpyBlockSize.x), 1, 1);

	// Imprimimos de ser el caso
	if (IMPRIMIR) { imprimir(h_grid); }

	// Ciclo principal de ejecucion
	t0 = static_cast<int>(clock());
	while (time < 1.0) {
		ghostRows <<< cpyGridRowsGridSize, cpyBlockSize >>> (dimFilas, dimColumnas, d_grid);
		ghostCols <<< cpyGridColsGridSize, cpyBlockSize >>> (dimFilas, dimColumnas, d_grid);
		if (GOLIF) {
			GOL_IF <<< gridSize, blockSize >>> (dimFilas, dimColumnas, d_grid, d_newGrid);
		}
		else {
			GOL <<< gridSize, blockSize >>> (dimFilas, dimColumnas, d_grid, d_newGrid);
		}
		// Intercambiamos punteros
		d_tmpGrid = d_grid;
		d_grid = d_newGrid;
		d_newGrid = d_tmpGrid;

		Noperaciones += N * M;

		t1 = static_cast<int>(clock());
		time = (double(t1 - t0) / CLOCKS_PER_SEC);
	}// Fin del ciclo principal de ejecucion

	// Pedimos los resultados de vuelta
	hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);

	// Imprimimos de ser el caso
	if (IMPRIMIR) {
		printf("\n");
		imprimir(h_grid);
	}

	// Imprimimos datos pedidos
	printf("Tiempo total: %f\n", time);
	printf("Numero de operaciones efectuadas %.0f\n", Noperaciones);

	// Se borra memoria
	hipFree(d_grid);
	hipFree(d_newGrid);
	free(h_grid);

	// Retorna main()
	return 0;

}

__global__
void GOL(int dimFilas, int dimColumnas, int *grid, int *newGrid) {

	// Queremos id en [1,dim]
	int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
	int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int id = iy * (dimColumnas + 2) + ix;

	int numNeighbors;
	if (iy <= dimFilas && ix <= dimColumnas) {

		// Obtenemos la cantidad de vecinos vivos
		numNeighbors = grid[id + (dimColumnas + 2)] + grid[id - (dimColumnas + 2)] //upper lower
			+ grid[id + 1] + grid[id - 1]             //right left
			+ grid[id + (dimColumnas + 3)] + grid[id - (dimColumnas + 3)] //diagonals
			+ grid[id - (dimColumnas + 1)] + grid[id + (dimColumnas + 1)];

		int cell = grid[id];

		// Ponemos las reglas del juego
		if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3)) {
			newGrid[id] = 1;
		}
		else if (cell == 0 && numNeighbors == 3) {
			newGrid[id] = 1;
		}
		else {
			newGrid[id] = cell;
		}

	}
}

__global__
void GOL_IF(int dimFilas, int dimColumnas, int *grid, int *newGrid) {

	// Queremos id ∈ [1,dim]
	int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
	int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int id = iy * (dimColumnas + 2) + ix;

	int numNeighbors = 0;

	if (iy <= dimFilas && ix <= dimColumnas) {

		// Obtenemos la cantidad de vecinos vivos
		if (grid[id + (dimColumnas + 2)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 2)]) { numNeighbors++; }
		if (grid[id + 1]) { numNeighbors++; }
		if (grid[id - 1]) { numNeighbors++; }
		if (grid[id + (dimColumnas + 3)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 3)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 1)]) { numNeighbors++; }
		if (grid[id + (dimColumnas + 1)]) { numNeighbors++; }

		int cell = grid[id];

		// Ponemos las reglas del juego
		if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3)) {
			newGrid[id] = 1;
		}
		else if (cell == 0 && numNeighbors == 3) {
			newGrid[id] = 1;
		}
		else {
			newGrid[id] = cell;
		}
	}

}

__global__ void ghostRows(int dimFilas, int dimColumnas, int *grid) {

	// Queremos id ∈ [1,dim]
	int id = blockDim.x * blockIdx.x + threadIdx.x + 1;

	if (id <= dimColumnas) {
		// Copiamos la primera fila real a la última fila
		grid[(dimColumnas + 2) * (dimFilas + 1) + id] = grid[(dimColumnas + 2) + id];

		// Copiamos la última fila real a la primera fila
		grid[id] = grid[(dimColumnas + 2) * dimFilas + id];
	}

}

__global__ void ghostCols(int dimFilas, int dimColumnas, int *grid) {

	// Queremos id ∈ [0,dim+1]
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id <= dimFilas + 1) {
		// Copia la primera columna real a la ultima
		grid[id * (dimColumnas + 2) + dimFilas + 1] = grid[id * (dimColumnas + 2) + 1];

		// Copia la última columna real a la primera
		grid[id * (dimColumnas + 2)] = grid[id * (dimColumnas + 2) + dimFilas];
	}

}

void imprimir(int *matriz) {
	for (int i = 1; i < N - 1; i++) {
		for (int j = 1; j < M - 1; j++) {
			printf("%d ", matriz[i * M + j]);
		}
		printf("\n");
	}
}