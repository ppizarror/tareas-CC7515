﻿/** Instrucciones
 *
 * El juego comienza con una configuracion al azar entre celdas vivas y muertas.
 *
 * Para modificar los valores de la ejecucion simplemente hay que modificar los
 * valores de las constantes declaradas mas abajo.
 *
 * N: Numero de filas que tendra la matriz que almacene el estado del juego.
 * M: Numero de columnas que tendra la matriz que almacene el estado del juego.
 *
 * BLOCK_SIZE: cantidad de threads que tendra cada bloque.
 * SRAND_VALUE: semilla que se ocupara para generar los numeros al azar.
 * GOLIF: Indicador en caso de que se quiera verificar la cantidad de celdas
 *        vecinas vivas usando solo IF's.
 * IMPRIMIR: Indicador en caso de que se necesite imprimir las matrices (esto
 *           afecta considerablemente el rendimiento de la solucion)
 */

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <ctime>
#include <fstream>

 /* Declaración de constantes */
#define SRAND_VALUE 1998	// Semilla para generar numeros random
#define IMPRIMIR 0			// Imprimir o no las matrices de entrada y de salida
#define T_LIMIT 1			// Tiempo límite de cálculo

/* Declaración de funciones */
__global__ void GOL(int dimFilas, int dimColumnas, int *grid, int *newGrid);

__global__ void ghostRows(int dimFilas, int dimColumnas, int *grid);

__global__ void ghostCols(int dimFilas, int dimColumnas, int *grid);

__global__ void GOL_IF(int dimFilas, int dimColumnas, int *grid, int *newGrid);

void imprimir(int *matriz, int n, int m);

/* Método principal */
int main(int argc, char *argv[]) {

	// Carga NxM desde un archivo
	std::ifstream infile;
	infile.open("NxM.txt");
	int x;
	int N = 0;
	int M = 0;
	int jfile = 0;
	while (infile >> x) {
		if (jfile == 0) { N = x; }
		else { M = x; }
		jfile = 1;
	}
	infile.close();

	// Carga IF, ejecutar el juego preguntando con IF (0:Falso 1:Verdadero)
	infile.open("IF.txt");
	int GOLIF = 0;
	while (infile >> x) {
		GOLIF = x;
	}
	infile.close();

	// Carga el tamaño de bloque
	infile.open("BLOCK_SIZE.txt");
	int BLOCK_SIZE = 0;
	while (infile >> x) {
		BLOCK_SIZE = x;
	}
	infile.close();

	printf("Cargando matriz %dx%d\n", N, M);
	printf("BLOCK SIZE: %d\n", BLOCK_SIZE);
	if (GOLIF) {
		printf("IF activado\n\n");
	}
	else {
		printf("IF desactivado\n\n");
	}

	int i, j;
	int *h_grid; // Matriz en CPU
	int *d_grid; // Matriz en GPU
	int *d_newGrid; // Matriz auxiliar usada solo en GPU
	int *d_tmpGrid; // Puntero auxiliar para cambiar las matrices

	signed t0, t1; // Variables para medir tiempo
	double time = 0; //variables para medir tiempo
	double Noperaciones = 0; // Variable para medir cantidad de operaciones ejecutadas

	int dimFilas = N; // Dimensiones del juego de la vida (Filas), sin contar las filas fantasmas
	int dimColumnas = M; // Dimensiones del juego de la vida (Columnas), sin contar las columnas fantasmas

	size_t bytes = sizeof(int) * (dimFilas + 2) *
		(dimColumnas + 2);// Se annade mas espacio para dejar filas y columnas fantasmas

	// Solicitamos memoria para la matriz en la CPU
	h_grid = (int *)malloc(bytes);

	// Solicitamos memoria para las matrices en la GPU
	hipMalloc(&d_grid, bytes);
	hipMalloc(&d_newGrid, bytes);

	// Colocamos valores aleatorios en la matriz inicialmente
	srand(SRAND_VALUE);
	for (i = 1; i <= dimFilas; i++) {
		for (j = 1; j <= dimColumnas; j++) {
			h_grid[i * (dimColumnas + 2) + j] = rand() % 2;
		}
	}

	// Copiamos valores iniciales de la matriz a la GPU
	hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_newGrid, h_grid, bytes, hipMemcpyHostToDevice);

	// Establecemos los tamannos de los bloques y la cantidad de bloques a utilizar
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
	int linGrid = (int)ceil((dimFilas * dimColumnas) / (float)(BLOCK_SIZE * BLOCK_SIZE));
	dim3 gridSize(linGrid, linGrid, 1);

	dim3 cpyBlockSize(BLOCK_SIZE, 1, 1);
	dim3 cpyGridRowsGridSize((int)ceil(dimFilas / (float)cpyBlockSize.x), 1, 1);
	dim3 cpyGridColsGridSize((int)ceil((dimColumnas + 2) / (float)cpyBlockSize.x), 1, 1);

	// Imprimimos de ser el caso
	if (IMPRIMIR) {
		imprimir(h_grid, N, M);
	}

	// Ciclo principal de ejecución
	t0 = static_cast<int>(clock());
	while (time < T_LIMIT) {
		ghostRows <<< cpyGridRowsGridSize, cpyBlockSize >>> (dimFilas, dimColumnas, d_grid);
		ghostCols <<< cpyGridColsGridSize, cpyBlockSize >>> (dimFilas, dimColumnas, d_grid);
		if (GOLIF) {
			GOL_IF <<< gridSize, blockSize >>> (dimFilas, dimColumnas, d_grid, d_newGrid);
		}
		else {
			GOL <<< gridSize, blockSize >>> (dimFilas, dimColumnas, d_grid, d_newGrid);
		}

		// Intercambiamos punteros
		d_tmpGrid = d_grid;
		d_grid = d_newGrid;
		d_newGrid = d_tmpGrid;

		Noperaciones += N * M;

		t1 = static_cast<int>(clock());
		time = (double(t1 - t0) / CLOCKS_PER_SEC);
	} // Fin del ciclo principal de ejecución

	// Pedimos los resultados de vuelta
	hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);

	// Imprimimos de ser el caso
	if (IMPRIMIR) {
		printf("\n");
		imprimir(h_grid, N, M);
	}

	// Imprimimos datos pedidos
	printf("Tiempo total: %f\n", time);
	printf("Numero de operaciones efectuadas: %.0f\n", Noperaciones);

	// Se borra memoria
	hipFree(d_grid);
	hipFree(d_newGrid);
	free(h_grid);

	// Retorna main()
	return 0;

}

__global__ void GOL(int dimFilas, int dimColumnas, int *grid, int *newGrid) {
	// Queremos id en [1,dim]
	int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
	int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int id = iy * (dimColumnas + 2) + ix;

	int numNeighbors;
	if (iy <= dimFilas && ix <= dimColumnas) {

		// Obtenemos la cantidad de vecinos vivos
		numNeighbors = grid[id + (dimColumnas + 2)] + grid[id - (dimColumnas + 2)] // upper lower
			+ grid[id + 1] + grid[id - 1] // right left
			+ grid[id + (dimColumnas + 3)] + grid[id - (dimColumnas + 3)] // diagonals
			+ grid[id - (dimColumnas + 1)] + grid[id + (dimColumnas + 1)];

		int cell = grid[id];

		// Ponemos las reglas del juego
		if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3)) {
			newGrid[id] = 1;
		}
		else if (cell == 0 && numNeighbors == 3) {
			newGrid[id] = 1;
		}
		else {
			newGrid[id] = cell;
		}
	}
}

__global__ void GOL_IF(int dimFilas, int dimColumnas, int *grid, int *newGrid) {
	// Queremos id en [1, dim]
	int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
	int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int id = iy * (dimColumnas + 2) + ix;

	int numNeighbors = 0;

	if (iy <= dimFilas && ix <= dimColumnas) {

		// Obtenemos la cantidad de vecinos vivos
		if (grid[id + (dimColumnas + 2)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 2)]) { numNeighbors++; }
		if (grid[id + 1]) { numNeighbors++; }
		if (grid[id - 1]) { numNeighbors++; }
		if (grid[id + (dimColumnas + 3)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 3)]) { numNeighbors++; }
		if (grid[id - (dimColumnas + 1)]) { numNeighbors++; }
		if (grid[id + (dimColumnas + 1)]) { numNeighbors++; }

		int cell = grid[id];

		// Ponemos las reglas del juego
		if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3)) {
			newGrid[id] = 1;
		}
		else if (cell == 0 && numNeighbors == 3) {
			newGrid[id] = 1;
		}
		else {
			newGrid[id] = cell;
		}
	}
}

__global__ void ghostRows(int dimFilas, int dimColumnas, int *grid) {
	// Queremos id en [1, dim]
	int id = blockDim.x * blockIdx.x + threadIdx.x + 1;
	if (id <= dimColumnas) {
		// Copiamos la primera fila real a la última fila
		grid[(dimColumnas + 2) * (dimFilas + 1) + id] = grid[(dimColumnas + 2) + id];
		// Copiamos la última fila real a la primera fila
		grid[id] = grid[(dimColumnas + 2) * dimFilas + id];
	}
}

__global__ void ghostCols(int dimFilas, int dimColumnas, int *grid) {
	// Queremos id en [0, dim+1]
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id <= dimFilas + 1) {
		// Copia la primera columna real a la ultima
		grid[id * (dimColumnas + 2) + dimFilas + 1] = grid[id * (dimColumnas + 2) + 1];
		// Copia la última columna real a la primera
		grid[id * (dimColumnas + 2)] = grid[id * (dimColumnas + 2) + dimFilas];
	}
}

void imprimir(int *matriz, int n, int m) {
	for (int i = 1; i < n - 1; i++) {
		for (int j = 1; j < m - 1; j++) {
			printf("%d ", matriz[i * m + j]);
		}
		printf("\n");
	}
}